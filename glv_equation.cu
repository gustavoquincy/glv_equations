#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <random>
#include <iomanip>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include "pcg_random.hpp"
//using pcg c++ implementation, pcg64, compilation requires -std=c++11 flag

typedef double_t value_type;
typedef thrust::host_vector< value_type > host_type;
typedef thrust::device_vector< value_type > state_type;
typedef runge_kutta_dopri5< state_type, value_type, state_type, value_type > stepper_type;

struct larger_than_zero
{   
    __host__ __device__
    bool operator()(const value_type x) { return x > 0; }
};

struct generalized_lotka_volterra_system
{
    generalized_lotka_volterra_system( size_t num_species ): m_num_species( num_species ) { }

    struct generalized_lotka_volterra_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t ) /* tuple t = { y, dydt, growth_rate, Sigma, interaction column } */
        {   
            thrust::device_vector<value_type> result( m_num_species );
            thrust::transform( y.begin(), y.end(), thrust::get<4>(t).begin(), result.begin(), thrust::multiplies<value_type>());
            thrust::device_vector<value_type> copy_result( m_num_species );
            thrust::fill( copy_result.begin(), copy_result.end(), 0);
            thrust::copy_if( result.begin(), result.end(), copy_result.begin(), larger_than_zero());
            value_type m_pos_sum = thrust::reduce( copy_result.begin(), copy_result.end(), 0.0 );
            thrust::fill( copy_result.begin(), copy_result.end(), 0);
            thrust::copy_if( result.begin(), result.end(), copy_result.begin(), !larger_than_zero());
            value_type m_neg_sum = thrust::reduce( copy_result.begin(), copy_result.end(), 0.0 );
            // steps above for derivation of m_pos_sum and m_neg_sum
            thrust::get<1>(t) = thrust::get<0>(t) * thrust::get<2>(t) * ( 1 + m_neg_sum + thrust::get<3>(t) * m_pos_sum / ( 1 + m_pos_sum )) - m_dilution * thrust::get<0>(t);
        }
    };


    void operator()( state_type& y , state_type& dydt, state_type growth_rate, state_type Sigma, state_type interaction )
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple( y.begin(), dydt.begin(), growth_rate.begin(), Sigma.begin(), interaction.begin() ) ),
                thrust::make_zip_iterator( thrust::make_tuple( y.end(), dydt.end(), growth_rate.end(), Sigma.end(), interaction.end() ) ),
                generalized_lotka_volterra_functor()
        );

    };
    
    state_type get_growth_rate() { return m_growth_rate; }

    void set_growth_rate( state_type growth_rate ) { thrust::copy( growth_rate.begin(), growth_rate.end(), m_growth_rate.begin() ); }

    value_type get_dilution() { return m_dilution; }

    void set_dilution( value_type dilution ) { thrust::copy( dilution.begin(), dilution.end(), m_dilution.begin() ); }

    value_type get_Sigma() { return m_Sigma; }

    void set_Sigma( state_type Sigma ) { thrust::copy( Sigma.begin(), Sigma.end(), m_Sigma.begin() ); }

    value_type get_interaction() { return m_interaction; }

    void set_interaction( matrix_type interaction ) { thrust::copy( interaction.begin(), interaction.end(), m_interaction.begin() ); }

};

// generator for random variable of uniform distribution U(a, b)
struct uniform_gen {
    uniform_gen(value_type a, value_type b): m_a(a), m_b(b) {}
    
    __host__
    value_type operator()() {
        pcg64 rng(pcg_extras::seed_seq_from<std::random_device{});
        // make a random number engine, use the 64-bit generator, 2^128 period, 2^127 streams
        std::uniform_real_distribution<double_t> uniform_dist(m_a, m_b);
        return uniform_dist(rng);
    }

    value_type m_a, m_b;
};

struct set_growthrate
{ 
    template<class Tuple >
    __host__
    void opeartor()( Tuple& t ) {
        thrust::get<3>(t) = thrust::get<0>(t) - thrust::get<1>(t) + 2 * thrust::get<1>(t) * thrust::get<2>(t); // t = { growth_rate_mean, growth_rate_width, unit_random_vec, growth_rate}
    }
};

struct is_below_promote_density
{   
    template<class Tuple >
    __host__
    bool operator()( Tuple t ) /* t = { 0 threshold_vector, 1 promote_dense, 2 compete_dense, 3 promote_mean, 4 promote_width, 5 compete_mean, 6 compete_width, 7 unit_random_vec, 8 interaction } (arity = 9)*/
    {
        return thrust::get<0>(t) <= thrust::get<1>(t);
    }
};

struct is_above_compete_density
{
    template<class Tuple >
    __host__
    bool opeartor()( Tuple t )
    {
        return thrust::get<0>(t) >= thrust::get<2>(t);
    }
};

struct set_promote_value
{
    template<class Tuple >
    __host__
    void operator()( Tuple& t )
    {
        thrust::get<8>(t) = thrust::get<3>(t) - thrust::get<4>(t) + 2 * thrust::get<4>(t) * thrust::get<7>(t);
    }
};

struct set_compete_value
{
    template<class Tuple >
    __host__
    void operator()( Tuple& t )
    {
        thrust::get<8>(t) = -1 * (thrust::get<5>(t) - thrust::get<6>(t) + 2 * thrust::get<6>(t) * thrust::get<7>(t));
    }
};

struct is_diagonal
{
    is_diagonal(size_t num_species): m_num_species(num_species) {}

    template<class Tuple >
    __host__
    bool operator()( Tuple t ) /* t = { index, interaction }*/
    {
        return thrust::get<0>(t) % (m_num_species + 1) == 1;
    }

    size_t m_num_species;
};

struct set_minus_one
{
    template<class Tuple >
    __host__
    void operator()( Tuple& t ) {
        thrust::get<1>(t) = -1.0;
    }
};

struct set_dilution
{
    set_dilution(value_type growth_rate_mean): m_growth_rate_mean(growth_rate_mean) {}

    __host__
    void operator()(value_type& di) {
        host_type random_vec_a(1), random_vec_b(1);
        thrust::generate(random_vec_a.begin(), random_vec_a.end(), uniform_gen(0, m_growth_rate_mean));
        thrust::generate(random_vec_b.begin(), random_vec_b.end(), uniform_gen(0, 0.3));
        di = random_vec_a[0] < random_vec_b[0] ? random_vec_a[0] :random_vec_b[0];
    }

    value_type m_growth_rate_mean;
};

struct normalize
{
    normalize(value_type normalized_by): m_normalized_by(normalized_by) {}
    
    __host__ __device__
    void operator()(value_type& x) {
        x /= m_normalized_by;
    }

    value_type m_normalized_by;
}

const size_t num_species = 10;
// initalize parameters, set the number of species to 10 in the generalized lv equation

const size_t outerloop = 200;  
// randomization for growth_rate, Sigma, interaction and dilution

const size_t innerloop = 500;
// randomization for initial condition of glv ODE

int main() {

    host_type growth_rate_host(num_species * outerloop)/* copy innerloop times */, Sigma_host(num_species * outerloop)/* copy innerloop times */, dilution_host(1 * outerloop) /* copy num_species*innerloop times */, interaction_host(num_species * num_species * outerloop) /* copy innerloop times */, initial_host(num_species * outerloop * innerloop);
    // outerloop - samplesize / innerloop - precision
    // randomize growth rate
    size_t dim = growth_rate_host.size();
    host_type growth_rate_mean(dim), growth_rate_width(dim), unit_random_vec(dim);
    host_type growth_rate_mean_host(1);
    thrust::generate(growth_rate_mean_host.begin(), growth_rate_mean_host.end(), uniform_gen(0.1, 1.5));
    thrust::fill(growth_rate_mean.begin(), groth_rate_mean.end(), growth_rate_mean_host[0]);
    host_type growth_rate_width_host(1);
    thrust::generate(growth_rate_width_host.begin(), growth_rate_width_host.end(), uniform_gen(0, growth_rate_mean_host[0]));
    thrust::fill(growth_rate_width.begin(), growth_rate_width.end(), growth_rate_width_host[0]);
    thrust::generate(unit_random_vec.begin(), unit_random_vec.end(), uniform_gen(0, 1.0));
    thrust::for_each( 
        thrust::make_zip_iterator( thrust::make_tuple( growth_rate_mean.begin(), growth_rate_width.begin(), unit_random_vec.begin(), growth_rate_host.begin() )),
        thrust::make_zip_iterator( thrust::make_tuple( growth_rate_mean.end(), growth_rate_width.end(), unit_random_vec.end(), growth_rate_host.end() )),
        set_growthrate() 
    );
    // randomize interaction
    size_t dim = interaction_host.size()
    host_type compete_dense(1), promote_dense(1);
    thrust::generate(compete_dense.begin(), compete_dense.end(), uniform_gen(0.5, 1.0));
    thrust::generate(promote_dense.begin(), promote_dense.end(), uniform_gen(0, 1 - compete_dense[0]));
    host_type promote_mean(dim), promote_width(dim), compete_mean(dim), compete_width(dim);
    host_type promote_mean_host(1), promote_width_host(1), compete_mean_host(1), compete_width_host(1);
    thrust::generate(compete_mean_host.begin(), compete_mean_host.end(), uniform_gen(0.5, 2.0)); 
    thrust::fill(compete_mean.begin(), compete_mean.end(), compete_mean_host[0]); 
    thrust::generate(promote_mean_host.begin(), promote_mean_host.end(), uniform_gen(0.01, 1.0));
    thrust::fill(promote_mean.begin(), promote_mean.end(), promote_mean_host[0]);
    thrust::generate(compete_width_host.begin(), compete_width_host.end(), uniform_gen(0, compete_mean_host[0]));
    thrust::fill(compete_width.begin(), compete_width.end(), compete_width_host[0]);
    thrust::generate(promote_width_host.begin(), promote_width_host.end(), uniform_gen(0, promote_mean_host[0]));
    thrust::fill(promote_width.begin(), promote_width.end(), promote_width_host[0]); 
    // generate once, then fill the device vector
    host_type threshold_vector(dim), unit_random_vec(dim);
    thrust::generate(threshold_vector.begin(), threshold_vector.end(), uniform_gen(0, 1.0));
    thrust::generate(unit_random_vec.begin(), unit_random_vec.end(), uniform_gen(0, 1.0));
    thrust::transform_if( 
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.begin(), promote_dense.begin(), compete_dense.begin(), promote_mean.begin(), promote_width.begin(), compete_mean.begin(), compete_width.begin(), unit_random_vec.begin(), interaction_host.begin() )),
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.end(), promote_dense.end(), compete_dense.end(), promote_mean.end(), promote_width.end(), compete_mean.end(), compete_width.end(), unit_random_vec.end(), interaction_host.end() )),
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.begin(), promote_dense.begin(), compete_dense.begin(), promote_mean.begin(), promote_width.begin(), compete_mean.begin(), compete_width.begin(), unit_random_vec.begin(), interaction_host.begin() )),
        is_below_promote_density(),
        set_promote_value() 
    );
    thrust::transform_if( 
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.begin(), promote_dense.begin(), compete_dense.begin(), promote_mean.begin(), promote_width.begin(), compete_mean.begin(), compete_width.begin(), unit_random_vec.begin(), interaction_host.begin() )),
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.end(), promote_dense.end(), compete_dense.end(), promote_mean.end(), promote_width.end(), compete_mean.end(), compete_width.end(), unit_random_vec.end(), interaction_host.end() )),
        thrust::make_zip_iterator( thrust::make_tuple( threshold_vector.begin(), promote_dense.begin(), compete_dense.begin(), promote_mean.begin(), promote_width.begin(), compete_mean.begin(), compete_width.begin(), unit_random_vec.begin(), interaction_host.begin() )),
        is_above_compete_density(),
        set_compete_value() 
    );
    thrust::host_vector<size_t> index(dim);
    thrust::sequence(index.begin(), index.end(), 1);
    thrust::transform_if( 
        thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction_host.begin() )), 
        thrust::make_zip_iterator( thrust::make_tuple( index.end(), interation_host.end() )), 
        thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction_host.begin() )), 
        set_minus_one(),
        is_diagonal(num_species) 
    );
    // randomize Sigma
    thrust::generate(Sigma_host.begin(), Sigma_host.end(), uniform_gen(0, 0.5));
    // randomize dilution
    thrust::for_each(dilution_host.begin(), dilution_host.end(), set_dilution(growth_rate_mean_host[0]));
    // randomize initial
    thrust::generate(initial_host.begin(), initial_host.end(), uniform_gen(0, 1.0));
    value_type initial_sum = thrust::reduce(initial_host.begin(), initial_host.end(), 0.0);
    thrust::for_each(initial_host.begin(), initial_host.end(), normalize(initial_sum));

    state_type growth_rate = growth_rate_host;
    state_type interaction = interaction_host;
    state_type Sigma = Sigma_host;
    state_type dilution = dilution_host;
    state_type initial = initial_host;
    // TODO: use hiprand to generate random numbers on device w/ hiprand kernel

    // TODO: solve ODE
    generalized_lotka_volterra_system glv( num_species );
    glv.set_growth_rate(growth_rate);
    glv.set_Sigma(Sigma);
    glv.set_interaction(interaction);
    glv.set_dilution(dilution);
    integrate_adaptive( make_controlled(1.0e-6, 1.0e-6, stepper_type()), glv, )

    // TODO: parse results with Euclidean distance aka 2-norm



    return 0;
}