#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <vector>
#include <cmath>
#include <iomanip>
#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust.hpp>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <arrow/api.h>
#include <arrow/csv/api.h>
#include <arrow/io/api.h>
#include <arrow/compute/api.h>
#include <parquet/arrow/writer.h>
#include <arrow/util/type_fwd.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <omp.h>

#pragma GCC diagnostic ignored "-Wunused-result"

using namespace boost::numeric::odeint;

typedef double_t value_type;
typedef thrust::host_vector< value_type > host_type;
typedef thrust::device_vector< value_type > state_type;

__global__ __launch_bound__(1024) void setup_kernel(hiprandState *state, int seed)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets device index seed, a different sequence number, no offset */
  hiprand_init(seed, id, 0, &state[id]);
}

__global__ __launch_bound__(1024) void initialize_parameters_growth_sigma(hiprandState *state, double_t *growth_rate, double_t *sigma, int sampleSize)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState localState = state[id];
// no dim 10**6
  if (id < sampleSize) {
    double_t growth_mean = 0.1 + 1.4 * hiprand_uniform_double(&localState);
    double_t growth_width = growth_mean * hiprand_uniform_double(&localstate);
    growth_rate[id] = growth_mean - growth_width + 2 * growth_width * hiprand_uniform_double(&localState);
    sigma[id] = 0.5 * hiprand_uniform_double(&localState);
  }
}

__global__ __launch_bound__(1024) void initialize_parameters_interaction(hiprandState *state, double_t *interaction, int sampleSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = state[id];
    if (id < sampleSize) {
        // n2o dim 10**8
        double_t compete_mean = 0.5 + 1.5 * hiprand_uniform_double(&localState);
        double_t compete_dense = 0.5 + 0.5 * hiprand_uniform_double(&localState);
        double_t promote_mean = 0.01 + 0.99 * hiprand_uniform_double(&localState);
        double_t promote_dense = ( 1 -  compete_dense ) * hiprand_uniform_double(&localState);
        double_t compete_width = compete_mean * hiprand_uniform_double(&localState);
        double_t promote_width = promote_mean * hiprand_uniform_double(&localState);
        (hiprand_uniform(&localState) <= promote_dense) ? 
            interaction[id] = promote_mean - promote_width + 2 * promote_width * hiprand_uniform_double(&localState) : (hiprand_uniform(&localState) >= compete_dense) ? 
            interaction[id] = -1 * (compete_mean - compete_width + 2 * compete_width * hiprand_uniform_double(&localState)) : 0 ;
    }
}

__global__ __launch_bound__(1024) void initialize_parameters_dilution(hiprandState *state, double_t *dilution, int sampleSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = state[id];
    if (id < sampleSize) {
        // i dim 10**3
        dilution[id] = std::min(growth_mean * hiprand_uniform_double(&localState), 0.3 * hiprand_uniform_double(&localState));
    }
}

__global__ __launch_bound__(1024) void initialize_initial(hiprandState *state, double_t *initial, int sampleSize)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = state[id];
    if (id < sampleSize) {
        // noi dim 10**9
        initial[id] = hiprand_uniform_double(&localState);
    }
}

struct generalized_lotka_volterra_system
{
    const size_t m_num_species, m_innerloop, m_outerloop;
    state_type m_growth_rate, m_Sigma, m_interaction, m_dilution; //pass-in value
    state_type growth_rate_i, Sigma_i, interaction_i, dilution_ni;  //operator-use value
    
    // m_growth_rate(num_species * outerloop)/* copy innerloop times */, m_Sigma(num_species * outerloop)/* copy innerloop times */, m_dilution(1 * outerloop) /* copy num_species*innerloop times */, m_interaction(num_species * num_species * outerloop) /* copy innerloop times */ 

    generalized_lotka_volterra_system( size_t num_species, size_t innerloop, size_t outerloop, state_type growth_rate, state_type Sigma, state_type interaction, state_type dilution )
    : m_num_species(num_species), m_innerloop(innerloop), m_outerloop(outerloop), m_growth_rate(growth_rate), m_Sigma(Sigma), m_interaction(interaction), m_dilution(dilution) {
        state_type growth_rate_i_scoped( m_growth_rate.size() * m_innerloop );
        state_type Sigma_i_scoped( m_Sigma.size() * m_innerloop );
        state_type dilution_i_scoped( m_dilution.size() * m_innerloop );
        state_type interaction_i_scoped( m_interaction.size() * m_innerloop );
        for (int i = 0; i < m_innerloop; ++i) {
            thrust::copy(m_growth_rate.begin(), m_growth_rate.end(), growth_rate_i_scoped.begin() + i * m_growth_rate.size());
            thrust::copy(m_Sigma.begin(), m_Sigma.end(), Sigma_i_scoped.begin() + i * m_Sigma.size());
            thrust::copy(m_dilution.begin(), m_dilution.end(), dilution_i_scoped.begin() + i * m_dilution.size());
            thrust::copy(m_interaction.begin(), m_interaction.end(), interaction_i_scoped.begin() + i * m_interaction.size());
        }
        growth_rate_i = growth_rate_i_scoped;
        Sigma_i = Sigma_i_scoped;
        interaction_i = interaction_i_scoped;
        state_type dilution_ni_scoped( dilution_i_scoped.size() * m_num_species );
        for (int i = 0; i < m_num_species; ++i) {
            thrust::copy(dilution_i_scoped.begin(), dilution_i_scoped.end(), dilution_ni_scoped.begin() + i * dilution_i_scoped.size());
        }
        dilution_ni = dilution_ni_scoped;
    }

    struct generalized_lotka_volterra_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )/* tuple t = { y, dydt, growth_rate, Sigma, dilution, pos_sum, neg_sum } (arity = 7)*/
        {   
            thrust::get<1>(t) = thrust::get<0>(t) * thrust::get<2>(t) * ( 1 + thrust::get<6>(t) + thrust::get<3>(t) * thrust::get<5>(t) / ( 1 + thrust::get<5>(t) )) - thrust::get<4>(t) * thrust::get<0>(t);
        }
    };

    void operator()( state_type& y , state_type& dydt, value_type t)
    {
        // copy y n times to make it n^2*io
        state_type y_n( y.size() * m_num_species );
        for (int i=0; i < m_num_species; ++i) {
            thrust::copy( y.begin(), y.end(), y_n.begin() + i * y.size() );
        }
        // multiply interaction with y piecewisely
        state_type result( interaction_i.size() );
        thrust::transform( y_n.begin(), y_n.end(), interaction_i.begin(), result.begin(), thrust::multiplies<value_type>() );
        // find pos_sum and neg_sum for every n in the result vector
        host_type result_host( result.size() );
        result_host = result;
        host_type pos_sum_host( m_num_species * m_innerloop * m_outerloop ), neg_sum_host( m_num_species * m_innerloop * m_outerloop );
        for (int i=0; i< m_num_species * m_innerloop * m_outerloop; ++i) {
            value_type pos_sum = 0.0;
            value_type neg_sum = 0.0;
            for (int j=0; j < m_num_species; ++j) {
                value_type vec_val = result_host[ i * m_num_species + j ];
                vec_val > 0 ? pos_sum += vec_val : neg_sum += vec_val;
            }
            pos_sum_host[i] = pos_sum;
            neg_sum_host[i] = neg_sum;
        }
        // then we have noi-dim pos_sum and noi-dim neg_sum
        state_type pos_sum( pos_sum_host.size() ), neg_sum( neg_sum_host.size() );
        pos_sum = pos_sum_host;
        neg_sum = neg_sum_host;

        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple( y.begin(), dydt.begin(), growth_rate_i.begin(), Sigma_i.begin(), dilution_ni.begin(), pos_sum.begin(), neg_sum.begin() ) ),
                thrust::make_zip_iterator( thrust::make_tuple( y.end(), dydt.end(), growth_rate_i.end(), Sigma_i.end(), dilution_ni.end(), pos_sum.end(), neg_sum.end() ) ),
                generalized_lotka_volterra_functor()
        );

        std::clog << "10 species abundance" << "\n";
        for (int i=0; i<y.size(); ++i) {
            std::clog << y[i] << std::endl;
            // store y
        }
        // write to arrow object once only
        std::clog << t << "\n";
    }

};

#pragma region
struct index_transform
{
    index_transform(size_t num_species): m_num_species(num_species) {
        m_counter = 0;
        m_i = 1;
    }

    __host__
    void operator()(size_t& idx)
    {
        bool is_diag = idx % (m_num_species + 1) == m_i;
        if ( is_diag ) m_counter += 1;
        if ( m_counter == m_num_species ) {
            m_i = (m_i + 1) % (m_num_species + 1);
            m_counter = 0;
        }
        idx = is_diag;
    }

    const size_t m_num_species;
    size_t m_i, m_counter;
};

struct set_minus_one
{
    template<class T >
    __host__ __device__
    T operator()( T t ) {
        thrust::get<1>(t) = -1.0;
        return t;
    }
};

struct normalize
{
    normalize(value_type normalized_by): m_normalized_by(normalized_by) {}
    
    __host__ __device__
    void operator()(value_type& x) {
        x /= m_normalized_by;
    }

    value_type m_normalized_by;
};

struct is_diagonal
{
    template<class T >
    __host__ __device__
    bool operator()(T t) /* t = { index, interaction }*/ {
        return thrust::get<0>(t);
    }
};
#pragma endregion

arrow::Status initial_condition_csv(double_t *growth_rate, double_t *Sigma, double_t *interaction, double_t *dilution , int64_t size) {
  arrow::DoubleBuilder doublebuilder;
  ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(in, size));
  std::shared_ptr<arrow::Array> random_number;
  ARROW_ASSIGN_OR_RAISE(random_number, doublebuilder.Finish());
  std::shared_ptr<arrow::ChunkedArray> random_number_chunks = std::make_shared<arrow::ChunkedArray>(random_number);
  std::shared_ptr<arrow::Field> field_random_number;
  std::shared_ptr<arrow::Schema> schema;
  field_random_number = arrow::field("random_number", arrow::float64());
  schema = arrow::schema({field_random_number});
  std::shared_ptr<arrow::Table> table = arrow::Table::Make(schema, {random_number_chunks});
  std::shared_ptr<arrow::io::FileOutputStream> outfile;
  ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open("test_out.csv"));
  ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, table->schema()));
  ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*table));
  ARROW_RETURN_NOT_OK(csv_writer->Close());

  return arrow::Status::OK();
}

const size_t num_species = 3; //10
// initalize parameters, set the number of species to 10 in the generalized lv equation

const size_t outerloop = 1000; //200  
// samplesize

const size_t innerloop = 200; //500
// precision

const unsigned int threadPerBlock = 1024;
const unsigned int blockCount = 207520; //the multiply is just larger than 8.5 * 10**8
const unsigned int totalThreads = threadPerblock * blockCount;

int main( int arc, char* argv[] ) 
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int sampleSize = num_species * outerloop;
    hiprandState *devStates;
    double_t *devResults, *hostResults, *devResults2, *hostResults2;
    hostResults = (double_t *)calloc(sampleSize * deviceCount, sizeof(double_t));
    #pragma omp parallel for num_threads(4) private(devResults, devResults2, devStates) shared(sampleSize, totalThreads, blockCount, threadPerBlock)
    for (int dev=0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipMalloc((void **)&devResults, sampleSize * sizeof(double_t));
        hipMalloc((void **)&devResults2, sampleSize * sizeof(double_t));
        hipMemset(devResults, 0, sampleSize * sizeof(double_t));
        hipMemset(devResults2, 0, sampleSize * sizeof(double_t));
        hipMalloc((void **)&devStates, totalThreads * sizeof(hiprandState));
        setup_kernel<<<blockCount, threadPerBlock>>>(devStates, dev);
        initialize_parameters_growth_sigma<<<blockCount, threadPerBlock>>>(devStates, devResults, devResults2, sampleSize);
        hipMemcpy(hostResults + dev * sampleSize, devResults, sampleSize * sizeof(double_t), hipMemcpyDeviceToHost);
        hipMemcpy(hostResults2 + dev * sampleSize, devResults2, sampleSize * sizeof(double_t), hipMemcpyDeviceToHost);
        hipFree(devResults);
        hipFree(devResults2);
    }
    state_type growth_rate(hostResults, hostResults +  sampleSize * deviceCount);
    state_type Sigma(hostResults2, hostResults2 + sampleSize * deviceCount);
    free(hostResults);
    free(hostResults2);

    sampleSize = num_species * num_species * outerloop;
    hostResults = (double_t *)calloc(sampleSize * deviceCount, sizeof(double_t));
    #pragma omp parallel for num_threads(4) private(devResults, devStates) shared(sampleSize, totalThreads, blockCount, threadPerBlock)
    for (int dev=0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipMalloc((void **)&devResults, sampleSize * sizeof(double_t));
        hipMemset(devResults, 0, sampleSize * sizeof(double_t));
        // hipMalloc((void **)&devStates, totalThreads * sizeof(hiprandState));
        // setup_kernel<<<blockCount, threadPerBlock>>>(devStates, dev);
        initialize_parameters_interaction<<<blockCount, threadPerBlock>>>(devStates, devResults, sampleSize);
        hipMemcpy(hostResults + dev * sampleSize, devResults, sampleSize * sizeof(double_t), hipMemcpyDeviceToHost);
        hipFree(devResults);
    }
    state_type interaction(hostResults, hostResults +  sampleSize * deviceCount);
    free(hostResults);
    thrust::host_vector<size_t> index_host(dim);
    thrust::sequence(index_host.begin(), index_host.end(), 1);
    thrust::for_each(index_host.begin(), index_host.end(), index_transform(num_species));
    state_type index = index_host;
    thrust::transform_if( 
        thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction.begin() )), 
        thrust::make_zip_iterator( thrust::make_tuple( index.end(), interaction.end() )), 
        thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction.begin() )), 
        set_minus_one(),
        is_diagonal() 
    );
    
    sampleSize = innerloop;
    hostResults = (double_t *)calloc(sampleSize * deviceCount, sizeof(double_t));
    #pragma omp parallel for num_threads(4) private(devResults, devStates) shared(sampleSize, totalThreads, blockCount, threadPerBlock)
    for (int dev=0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipMalloc((void **)&devResults, sampleSize * sizeof(double_t));
        hipMemset(devResults, 0, sampleSize * sizeof(double_t));
        intialize_parameters_dilution<<<blockCount, threadPerBlock>>>(devStates, devResults, sampleSize);
        hipMemcpy(hostResults + dev * sampleSize, devResults, sampleSize * sizeof(double_t), hipMemcpyDeviceToHost);
        hipFree(devResults);
    }
    state_type dilution(hostResults, hostResults +  sampleSize * deviceCount);
    free(hostResults);
    
    sampleSize = num_species * innerloop * outerloop;
    hostResults = (double_t *)calloc(sampleSize * deviceCount, sizeof(double_t));
    #pragma omp parallel for num_threads(4) private(devResults, devStates) shared(sampleSize, totalThreads, blockCount, threadPerBlock)
    for (int dev=0; dev < deviceCount; ++dev) {
        hipSetDevice(dev);
        hipMalloc((void **)&devResults, sampleSize * sizeof(double_t));
        hipMemset(devResults, 0, sampleSize * sizeof(double_t));
        initialize_initial<<<blockCount, threadPerBlock>>>(devStates, devResults, sampleSize);
        hipMemcpy(hostResults + dev * sampleSize, devResults, sampleSize * sizeof(double_t), hipMemcpyDeviceToHost);
        hipFree(devResults);
    }
    state_type initial(hostResults, hostResults +  sampleSize * deviceCount);
    for (int i = 0; i < innerloop * outerloop; ++i ) {
        double_t sum = thrust::reduce(initial.begin() + i * num_species, initial.begin() + (i + 1) * num_species, 0);
        thrust::for_each(initial.begin() + i * num_species, initial.begin() + (i + 1) * num_species, normalize(sum));
    }

    typedef runge_kutta_dopri5< state_type , value_type , state_type , value_type > stepper_type;
    generalized_lotka_volterra_system glv_system( num_species, innerloop, outerloop, growth_rate, Sigma, interaction, dilution );

    integrate_adaptive( make_dense_output(1.0e-6, 1.0e-6, stepper_type() ), glv_system, initial , 0.0, 1.0, 0.01);

    // TODO: parse results with Euclidean distance aka 2-norm
    

    return 0;
}