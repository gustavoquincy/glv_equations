#include "hip/hip_runtime.h"
    #include <iostream>
    #include <cstdlib>
    #include <vector>
    #include <cmath>
    #include <iomanip>
    #include <boost/numeric/odeint.hpp>
    #include <boost/numeric/odeint/external/thrust/thrust.hpp>
    #include <thrust/device_vector.h>
    #include <thrust/host_vector.h>
    #include <thrust/reduce.h>
    #include <thrust/functional.h>
    #include <thrust/fill.h>
    #include <thrust/for_each.h>
    #include <thrust/generate.h>
    #include <thrust/reduce.h>
    #include <thrust/sequence.h>
    #include <arrow/api.h>
    #include <arrow/csv/api.h>
    #include <arrow/io/api.h>
    #include <arrow/ipc/api.h>
    #include <arrow/compute/api.h>
    #include <parquet/arrow/writer.h>
    #include <arrow/util/type_fwd.h>
    #include <stdio.h>
    #include <stdlib.h>
    #include <hip/hip_runtime.h>
    #include <hiprand/hiprand_kernel.h>
    #include <hiprand.h>
    #include <omp.h>

    #pragma GCC diagnostic ignored "-Wunused-result"

    using namespace boost::numeric::odeint;

    typedef double_t value_type;
    typedef thrust::host_vector< value_type > host_type;
    typedef thrust::device_vector< value_type > state_type;

    #pragma region //hiprand kernels

    __global__ __launch_bounds__(1024) void initialize_parameters_growth_sigma(hiprandState *state, double_t *growth_rate, double_t *sigma, int sampleSize, int dev, int offset, double_t growth_mean)
    {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(dev, id, offset, &state[id]);
    // no dim 10**6
    if (id < sampleSize) {
        double_t growth_width = growth_mean * hiprand_uniform_double(&state[id]);
        growth_rate[id] = growth_mean - growth_width + 2 * growth_width * hiprand_uniform_double(&state[id]);
        sigma[id] = 0.5 * hiprand_uniform_double(&state[id]);
    }
    }

    __global__ __launch_bounds__(1024) void initialize_parameters_interaction(hiprandState *state, double_t *interaction, int sampleSize, int dev, int offset)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(dev, id, offset, &state[id]);
        if (id < sampleSize) {
            // n2o dim 10**8
            double_t compete_mean = 0.5 + 1.5 * hiprand_uniform_double(&state[id]);
            double_t compete_dense = 0.5 + 0.5 * hiprand_uniform_double(&state[id]);
            double_t promote_mean = 0.01 + 0.99 * hiprand_uniform_double(&state[id]);
            double_t promote_dense = ( 1 -  compete_dense ) * hiprand_uniform_double(&state[id]);
            double_t compete_width = compete_mean * hiprand_uniform_double(&state[id]);
            double_t promote_width = promote_mean * hiprand_uniform_double(&state[id]);
            (hiprand_uniform(&state[id]) <= promote_dense) ? 
                interaction[id] = promote_mean - promote_width + 2 * promote_width * hiprand_uniform_double(&state[id]) : (hiprand_uniform(&state[id]) >= compete_dense) ? 
                interaction[id] = -1 * (compete_mean - compete_width + 2 * compete_width * hiprand_uniform_double(&state[id])) : 0 ;
        }
    }

    __global__ __launch_bounds__(1024) void initialize_parameters_dilution(hiprandState *state, double_t *dilution, int sampleSize, int dev, int offset, double_t growth_mean)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(dev, id, offset, &state[id]);
        if (id < sampleSize) {
            // i dim 10**3
            dilution[id] = std::min(growth_mean * hiprand_uniform_double(&state[id]), 0.3 * hiprand_uniform_double(&state[id]));
        }
    }

    __global__ __launch_bounds__(1024) void initialize_initial(hiprandState *state, double_t *initial, int sampleSize, int dev, int offset)
    {
        int id = threadIdx.x + blockIdx.x * blockDim.x;
        hiprand_init(dev, id, offset, &state[id]);
        if (id < sampleSize) {
            // noi dim 10**9
            initial[id] = hiprand_uniform_double(&state[id]);
        }
    }
    #pragma endregion

    arrow::Status state_write_table(double_t *state, int n, int o, int i, double_t time) {
        arrow::DoubleBuilder doublebuilder;
        ARROW_RETURN_NOT_OK(doublebuilder.AppendNulls(n * i)); // ni, the column length of the table
        ARROW_ASSIGN_OR_RAISE(std::shared_ptr<arrow::Array> arr_null, doublebuilder.Finish());
        std::shared_ptr<arrow::Table> state_table = arrow::Table::Make(arrow::schema({arrow::field("", arrow::float64())}), {arr_null});
        for (int j = 0; j < o; ++j){
            for (int k = 0; k < i; ++k) {
                double_t *p_start = state + n * j + n * o * k;
                double_t *p_end = state + n * j + n * o * k + n;
                for (double_t *ptr = p_start; ptr < p_end; ++ptr) {
                    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(ptr, 1));
                }
            }
            ARROW_ASSIGN_OR_RAISE(std::shared_ptr<arrow::Array> sys_timesnap, doublebuilder.Finish());
            std::shared_ptr<arrow::ChunkedArray> sys_timesnap_chunks = std::make_shared<arrow::ChunkedArray>(sys_timesnap);
            std::string str = "system_" + std::to_string(j);
            std::shared_ptr<arrow::Field> field = arrow::field(str, arrow::float64());
            ARROW_ASSIGN_OR_RAISE(state_table, state_table->AddColumn(1, field, sys_timesnap_chunks));
        } 
        ARROW_ASSIGN_OR_RAISE(state_table, state_table->RemoveColumn(0)); // remove the null column, aka the first column
        std::shared_ptr<arrow::io::FileOutputStream> outfile;
        std::string str = "system_state_at_time_" + std::to_string(time) + ".csv";
        ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open(str));
        ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, state_table->schema()));
        ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*state_table));
        ARROW_RETURN_NOT_OK(csv_writer->Close());
        return arrow::Status::OK();
    }

    struct generalized_lotka_volterra_system
    {
        const size_t m_num_species, m_innerloop, m_outerloop;
        state_type m_growth_rate, m_Sigma, m_interaction, m_dilution; //pass-in value
        state_type growth_rate_i, Sigma_i, interaction_i, dilution_ni;  //operator-use value
        
        // m_growth_rate(no), m_Sigma(no), m_dilution(o), m_interaction(nno)

        generalized_lotka_volterra_system( size_t num_species, size_t innerloop, size_t outerloop, state_type growth_rate, state_type Sigma, state_type interaction, state_type dilution )
        : m_num_species(num_species), m_innerloop(innerloop), m_outerloop(outerloop), m_growth_rate(growth_rate), m_Sigma(Sigma), m_interaction(interaction), m_dilution(dilution) {
            state_type growth_rate_i_scoped( m_growth_rate.size() * m_innerloop );
            state_type Sigma_i_scoped( m_Sigma.size() * m_innerloop );
            state_type dilution_i_scoped( m_dilution.size() * m_innerloop );
            state_type interaction_i_scoped( m_interaction.size() * m_innerloop );
            for (int i = 0; i < m_innerloop; ++i) {
                thrust::copy(m_growth_rate.begin(), m_growth_rate.end(), growth_rate_i_scoped.begin() + i * m_growth_rate.size());
                thrust::copy(m_Sigma.begin(), m_Sigma.end(), Sigma_i_scoped.begin() + i * m_Sigma.size());
                thrust::copy(m_dilution.begin(), m_dilution.end(), dilution_i_scoped.begin() + i * m_dilution.size());
                thrust::copy(m_interaction.begin(), m_interaction.end(), interaction_i_scoped.begin() + i * m_interaction.size());
            }
            growth_rate_i = growth_rate_i_scoped;
            Sigma_i = Sigma_i_scoped;
            interaction_i = interaction_i_scoped;
            state_type dilution_ni_scoped( dilution_i_scoped.size() * m_num_species );
            for (int i = 0; i < m_num_species; ++i) {
                thrust::copy(dilution_i_scoped.begin(), dilution_i_scoped.end(), dilution_ni_scoped.begin() + i * dilution_i_scoped.size());
            }
            dilution_ni = dilution_ni_scoped;
        }

        struct generalized_lotka_volterra_functor
        {
            template< class Tuple >
            __host__ __device__
            void operator()( Tuple t )/* tuple t = { y, dydt, growth_rate, Sigma, dilution, pos_sum, neg_sum } (arity = 7)*/
            {   
                thrust::get<1>(t) = thrust::get<0>(t) * thrust::get<2>(t) * ( 1 + thrust::get<6>(t) + 0.0 * thrust::get<3>(t) * thrust::get<5>(t) / ( 1 + thrust::get<5>(t) )) - thrust::get<4>(t) * thrust::get<0>(t);
            }
        };

        void operator()( state_type& y , state_type& dydt, value_type t)
        {
            // copy y n times to make it n^2*io
            state_type y_n( y.size() * m_num_species );
            for (int i=0; i <  m_innerloop * m_outerloop; ++i) {
                for (int j=0; j < m_num_species; ++j) {
                        thrust::copy( y.begin() + i * m_num_species, y.begin() + (i + 1) * m_num_species, y_n.begin() + i * m_num_species * m_num_species + j * m_num_species );
                }
            }
            // multiply interaction with y piecewisely
            state_type result( interaction_i.size() );
            thrust::transform( y_n.begin(), y_n.end(), interaction_i.begin(), result.begin(), thrust::multiplies<value_type>() );
            // find pos_sum and neg_sum for every n in the result vector
            host_type result_host( result.size() );
            result_host = result;
            host_type pos_sum_host( m_num_species * m_innerloop * m_outerloop ), neg_sum_host( m_num_species * m_innerloop * m_outerloop );
            for (int i=0; i < m_num_species * m_innerloop * m_outerloop; ++i) {
                value_type pos_sum = 0.0;
                value_type neg_sum = 0.0;
                for (int j=0; j < m_num_species; ++j) {
                    value_type vec_val = result_host[ i * m_num_species + j ];
                    vec_val > 0 ? pos_sum += vec_val : neg_sum += vec_val;
                }
                // std::clog << "print pos_sum: " << pos_sum << std::endl;
                // std::clog << "print neg_sum: " << neg_sum << std::endl;
                pos_sum_host[i] = pos_sum;
                neg_sum_host[i] = neg_sum;
            }
            // then we have noi-dim pos_sum and noi-dim neg_sum
            state_type pos_sum( pos_sum_host.size() ), neg_sum( neg_sum_host.size() );
            pos_sum = pos_sum_host;
            neg_sum = neg_sum_host;

            thrust::for_each(
                    thrust::make_zip_iterator( thrust::make_tuple( y.begin(), dydt.begin(), growth_rate_i.begin(), Sigma_i.begin(), dilution_ni.begin(), pos_sum.begin(), neg_sum.begin() ) ),
                    thrust::make_zip_iterator( thrust::make_tuple( y.end(), dydt.end(), growth_rate_i.end(), Sigma_i.end(), dilution_ni.end(), pos_sum.end(), neg_sum.end() ) ),
                    generalized_lotka_volterra_functor()
            );

            double_t *raw_y = thrust::raw_pointer_cast(y.data());
            arrow::Status status = state_write_table(raw_y, m_num_species, m_outerloop, m_innerloop, t);
            if (!status.ok()) {
                std::clog << status.ToString() << std::endl;
            }
        }

    };

    #pragma region //write table arrow status
    arrow::Status growth_rate_sigma_write_table(double_t *growth_rate, double_t *Sigma, int64_t size) {
    arrow::DoubleBuilder doublebuilder;
    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(growth_rate, size));
    std::shared_ptr<arrow::Array> growth_rate_arr;
    ARROW_ASSIGN_OR_RAISE(growth_rate_arr, doublebuilder.Finish());
    std::shared_ptr<arrow::ChunkedArray> growth_rate_chunks = std::make_shared<arrow::ChunkedArray>(growth_rate_arr);
    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(Sigma, size));
    std::shared_ptr<arrow::Array> sigma_arr;
    ARROW_ASSIGN_OR_RAISE(sigma_arr, doublebuilder.Finish());
    std::shared_ptr<arrow::ChunkedArray> sigma_chunks = std::make_shared<arrow::ChunkedArray>(sigma_arr);
    std::shared_ptr<arrow::Field> field_growth_rate, field_sigma;
    std::shared_ptr<arrow::Schema> schema;
    field_growth_rate = arrow::field("growth rate", arrow::float64());
    field_sigma = arrow::field("sigma", arrow::float64());
    schema = arrow::schema({field_growth_rate, field_sigma});
    std::shared_ptr<arrow::Table> table = arrow::Table::Make(schema, {growth_rate_chunks, sigma_chunks});
    std::shared_ptr<arrow::io::FileOutputStream> outfile;
    ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open("growth_rate_and_sigma.csv"));
    ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, table->schema()));
    ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*table));
    ARROW_RETURN_NOT_OK(csv_writer->Close());
    return arrow::Status::OK();
    }

    arrow::Status interaction_write_table(double_t *interaction, int64_t size) {
    arrow::DoubleBuilder doublebuilder;
    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(interaction, size));
    std::shared_ptr<arrow::Array> interaction_arr;
    ARROW_ASSIGN_OR_RAISE(interaction_arr, doublebuilder.Finish());
    std::shared_ptr<arrow::ChunkedArray> interaction_chunks = std::make_shared<arrow::ChunkedArray>(interaction_arr);
    std::shared_ptr<arrow::Field> field_interaction;
    std::shared_ptr<arrow::Schema> schema;
    field_interaction = arrow::field("interaction matrix", arrow::float64());
    schema = arrow::schema({field_interaction});
    std::shared_ptr<arrow::Table> table = arrow::Table::Make(schema, {interaction_chunks});
    std::shared_ptr<arrow::io::FileOutputStream> outfile;
    ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open("interaction.csv"));
    ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, table->schema()));
    ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*table));
    ARROW_RETURN_NOT_OK(csv_writer->Close());
    return arrow::Status::OK();
    }

    arrow::Status dilution_write_table(double_t *dilution, int64_t size) {
    arrow::DoubleBuilder doublebuilder;
    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(dilution, size));
    std::shared_ptr<arrow::Array> dilution_arr;
    ARROW_ASSIGN_OR_RAISE(dilution_arr, doublebuilder.Finish());
    std::shared_ptr<arrow::ChunkedArray> dilution_chunks = std::make_shared<arrow::ChunkedArray>(dilution_arr);
    std::shared_ptr<arrow::Field> field_dilution;
    std::shared_ptr<arrow::Schema> schema;
    field_dilution = arrow::field("dilution", arrow::float64());
    schema = arrow::schema({field_dilution});
    std::shared_ptr<arrow::Table> table = arrow::Table::Make(schema, {dilution_chunks});
    std::shared_ptr<arrow::io::FileOutputStream> outfile;
    ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open("dilution.csv"));
    ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, table->schema()));
    ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*table));
    ARROW_RETURN_NOT_OK(csv_writer->Close());
    return arrow::Status::OK();
    }
    arrow::Status initial_write_table(double_t *initial, int64_t size) {
    arrow::DoubleBuilder doublebuilder;
    ARROW_RETURN_NOT_OK(doublebuilder.AppendValues(initial, size));
    std::shared_ptr<arrow::Array> initial_arr;
    ARROW_ASSIGN_OR_RAISE(initial_arr, doublebuilder.Finish());
    std::shared_ptr<arrow::ChunkedArray> initial_chunks = std::make_shared<arrow::ChunkedArray>(initial_arr);
    std::shared_ptr<arrow::Field> field_initial;
    std::shared_ptr<arrow::Schema> schema;
    field_initial = arrow::field("initial", arrow::float64());
    schema = arrow::schema({field_initial});
    std::shared_ptr<arrow::Table> table = arrow::Table::Make(schema, {initial_chunks});
    std::shared_ptr<arrow::io::FileOutputStream> outfile;
    ARROW_ASSIGN_OR_RAISE(outfile, arrow::io::FileOutputStream::Open("initial.csv"));
    ARROW_ASSIGN_OR_RAISE(auto csv_writer, arrow::csv::MakeCSVWriter(outfile, table->schema()));
    ARROW_RETURN_NOT_OK(csv_writer->WriteTable(*table));
    ARROW_RETURN_NOT_OK(csv_writer->Close());
    return arrow::Status::OK();
    }
    #pragma endregion

    #pragma region //functor for thrust vector interaction and initial
    struct index_transform
    {
        index_transform(size_t num_species): m_num_species(num_species) {
            m_counter = 0;
            m_i = 1;
        }

        __host__
        void operator()(size_t& idx)
        {
            bool is_diag = idx % (m_num_species + 1) == m_i;
            if ( is_diag ) m_counter += 1;
            if ( m_counter == m_num_species ) {
                m_i = (m_i + 1) % (m_num_species + 1);
                m_counter = 0;
            }
            idx = is_diag;
        }

        const size_t m_num_species;
        size_t m_i, m_counter;
    };

    struct set_minus_one
    {
        template<class T >
        __host__ __device__
        T operator()( T t ) {
            thrust::get<1>(t) = -1.0;
            return t;
        }
    };

    struct normalize
    {
        normalize(value_type normalized_by): m_normalized_by(normalized_by) {}
        
        __host__ __device__
        void operator()(value_type& x) {
            x /= m_normalized_by;
        }

        value_type m_normalized_by;
    };

    struct is_diagonal
    {
        template<class T >
        __host__ __device__
        bool operator()(T t) /* t = { index, interaction }*/ {
            return thrust::get<0>(t);
        }
    };
    #pragma endregion

    const size_t num_species = 3; //10

    const size_t outerloop = 200; //1000  

    const size_t innerloop = 200; //500

    const unsigned int threadPerBlock = 1024;
    const unsigned int blockCount = 207520; //the multiply is just larger than 8.5 * 10**8
    const unsigned int totalThreads = threadPerBlock * blockCount;

    int main( int arc, char* argv[] ) 
    {
        int deviceCount;
        hipGetDeviceCount(&deviceCount);

        int noSize = num_species * outerloop / deviceCount;
        int nnoSize = num_species * num_species * outerloop / deviceCount;
        int oSize = outerloop / deviceCount;
        int noiSize = num_species * outerloop * innerloop / deviceCount;
        hiprandState *devStates;
        double_t *growth_rate_host, *growth_rate_dev, *sigma_host, *sigma_dev, *interaction_host, *interaction_dev, *dilution_host, *dilution_dev, *initial_host, *initial_dev;
        growth_rate_host = (double_t *)calloc(noSize * deviceCount, sizeof(double_t));
        sigma_host = (double_t *)calloc(noSize * deviceCount, sizeof(double_t));
        //interaction_host = (double_t *)calloc(nnoSize * deviceCount, sizeof(double_t));
        dilution_host = (double_t *)calloc(oSize * deviceCount, sizeof(double_t));
        initial_host = (double_t *)calloc(noiSize * deviceCount, sizeof(double_t));
        double_t random = (double_t)rand() / RAND_MAX;
        double_t growth_mean = 0.1 + 1.4 * random;
        #pragma omp parallel for num_threads(4) private(devStates, growth_rate_dev, sigma_dev, interaction_dev, dilution_dev, initial_dev) shared(totalThreads, blockCount, threadPerBlock, noSize, nnoSize, oSize, noiSize)
        for (int dev=0; dev < deviceCount; ++dev) {
            hipSetDevice(dev);
            hipMalloc((void **)&growth_rate_dev, noSize * sizeof(double_t));
            hipMalloc((void **)&sigma_dev, noSize * sizeof(double_t));
            hipMemset(growth_rate_dev, 0, noSize * sizeof(double_t));
            hipMemset(sigma_dev, 0, noSize * sizeof(double_t));
            hipMalloc((void **)&devStates, totalThreads * sizeof(hiprandState));
            initialize_parameters_growth_sigma<<<blockCount, threadPerBlock>>>(devStates, growth_rate_dev, sigma_dev, noSize, dev, 0, growth_mean);
            hipMemcpy(growth_rate_host + dev * noSize, growth_rate_dev, noSize * sizeof(double_t), hipMemcpyDeviceToHost);
            hipMemcpy(sigma_host + dev * noSize, sigma_dev, noSize * sizeof(double_t), hipMemcpyDeviceToHost);
            hipFree(growth_rate_host);
            hipFree(sigma_host);
            /*
            hipMalloc((void **)&interaction_dev, nnoSize * sizeof(double_t));
            hipMemset(interaction_dev, 0, nnoSize * sizeof(double_t));
            initialize_parameters_interaction<<<blockCount, threadPerBlock>>>(devStates, interaction_dev, nnoSize, dev, 1);
            hipMemcpy(interaction_host + dev * nnoSize, interaction_dev, nnoSize * sizeof(double_t), hipMemcpyDeviceToHost);
            hipFree(interaction_dev);
            */
            hipMalloc((void **)&dilution_dev, oSize * sizeof(double_t));
            hipMemset(dilution_dev, 0, oSize * sizeof(double_t));
            initialize_parameters_dilution<<<blockCount, threadPerBlock>>>(devStates, dilution_dev, oSize, dev, 2, growth_mean);
            hipMemcpy(dilution_host + dev * oSize, dilution_dev, oSize * sizeof(double_t), hipMemcpyDeviceToHost);
            hipFree(dilution_dev);
            hipMalloc((void **)&initial_dev, noiSize * sizeof(double_t));
            hipMemset(initial_dev, 0, noiSize * sizeof(double_t));
            initialize_initial<<<blockCount, threadPerBlock>>>(devStates, initial_dev, noiSize, dev, 3);
            hipMemcpy(initial_host + dev * noiSize, initial_dev, noiSize * sizeof(double_t), hipMemcpyDeviceToHost);
            hipFree(initial_dev);
        }
        state_type growth_rate(growth_rate_host, growth_rate_host +  noSize * deviceCount);
        state_type Sigma(sigma_host, sigma_host + noSize * deviceCount);
        free(growth_rate_host);
        free(sigma_host);
        /*
        state_type interaction(interaction_host, interaction_host +  nnoSize * deviceCount);
        free(interaction_host);
        */
        state_type interaction(nnoSize * deviceCount);
        size_t dim = interaction.size();
        thrust::host_vector<size_t> index_host(dim);
        thrust::sequence(index_host.begin(), index_host.end(), 1);
        thrust::for_each(index_host.begin(), index_host.end(), index_transform(num_species));
        state_type index = index_host;
        thrust::transform_if( 
            thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction.begin() )), 
            thrust::make_zip_iterator( thrust::make_tuple( index.end(), interaction.end() )), 
            thrust::make_zip_iterator( thrust::make_tuple( index.begin(), interaction.begin() )), 
            set_minus_one(),
            is_diagonal() 
        );
        state_type dilution(dilution_host, dilution_host +  oSize * deviceCount);
        free(dilution_host);
        state_type initial(initial_host, initial_host +  noiSize * deviceCount);
        free(initial_host);
        for (int i = 0; i < innerloop * outerloop - 1; ++i ) {
            double_t sum = thrust::reduce(initial.begin() + i * num_species, initial.begin() + (i + 1) * num_species, 0.0);
            thrust::for_each(initial.begin() + i * num_species, initial.begin() + (i + 1) * num_species, normalize(sum));
        }

        int64_t size = growth_rate.size();
        double_t *raw_growth_rate = thrust::raw_pointer_cast(growth_rate.data());
        double_t *raw_sigma = thrust::raw_pointer_cast(Sigma.data());
        growth_rate_sigma_write_table(raw_growth_rate, raw_sigma, size);
        double_t *raw_interaction = thrust::raw_pointer_cast(interaction.data());
        size = interaction.size();
        interaction_write_table(raw_interaction, size);
        double_t *raw_dilution = thrust::raw_pointer_cast(dilution.data());
        size = dilution.size();
        dilution_write_table(raw_dilution, size);
        size = initial.size();
        double_t *raw_initial = thrust::raw_pointer_cast(initial.data());
        initial_write_table(raw_initial, size);

        typedef runge_kutta_dopri5< state_type , value_type , state_type , value_type > stepper_type;
        generalized_lotka_volterra_system glv_system( num_species, innerloop, outerloop, growth_rate/*no*/, Sigma/*no*/, interaction/*nno*/, dilution/*o*/);

        integrate_adaptive( make_dense_output(1.0e-6, 1.0e-6, stepper_type() ), glv_system, initial/*noi*/ , 0.0, 100.0, 0.1);

        // TODO: parse results with Euclidean distance aka 2-norm
        

        return 0;
    }